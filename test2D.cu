#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

__device__ int taskIdx;         // "slateIdx"



// verify ID of SM
__device__ __inline__ uint32_t get_smid(){
    uint32_t smid;
    asm volatile("mov.u32 %0, %%smid;" : "=r"(smid));
    return smid;
}

__global__ void TransformedKernel(  int sm_low, int sm_high,
                                    int* g_data, int inc_value,
                                    int grid_size,
                                    int *block_index,  int *max_blocks)
{
    __shared__ int smid;
    __shared__ bool valid;
    __shared__ int globIdx;

    __shared__ int logicalBlockIdx;
    __shared__ int physicalBlockIdx;
    __shared__ uint3 shared_blockID;
    
    const int leader = ( threadIdx.x == 0 &&
                         threadIdx.y == 0 &&
                         threadIdx.z == 0);

    if(leader){
        // logicalBlockIdx initialization
        logicalBlockIdx = 0;
        smid = get_smid();

        valid = !( smid < sm_low ||
                   smid > sm_high );
    }
    __syncthreads();
    
    if(!valid)
        return;

    int range = sm_high - sm_low + 1;               // number of PERSISTENT WORKERS
   
    if(leader)
    {
        physicalBlockIdx = atomicAdd(&(block_index[smid]), 1);
    }
    __syncthreads();
    

    while(1)
    {
        while(physicalBlockIdx >= *max_blocks)
        {
            physicalBlockIdx = block_index[smid];
        }

        if(leader)
        {
            globIdx = atomicAdd(&taskIdx, 1);
            
            logicalBlockIdx = globIdx + range;      // next index which PERSISTENT WORKER will have
            
            // calculate real block index which the user specified 
            //           from the modified index "globIdx" of transformed Grid K*
            shared_blockID.x = globIdx % gridDim.x -1;
            shared_blockID.y = globIdx / gridDim.x;
        }

        __syncthreads();
        
        // assume this case can have 1-D or 2-D grid like "slate"
        uint3 blockID = { shared_blockID.x, shared_blockID.y, 1 };
        
        // assume this case have 2D block
        int block_idx = blockID.y * gridDim.x + blockID.x;
        int thread_idx = block_idx * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;
        
        // original kernel
        g_data[thread_idx] = g_data[thread_idx] + inc_value;

        if(leader)
            atomicSub( &(block_index[smid]),1);
        
        if(logicalBlockIdx >= grid_size)
            break;
    }
}


// original test kernel for 2D block
__global__ void incremental_kernel_2d(int* g_data, int inc_value)
{
    int block_idx  = blockIdx.y * gridDim.x + blockIdx.x;
    int thread_idx = block_idx * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;
    g_data[thread_idx] = g_data[thread_idx] + inc_value;
}


// check the correctness of kernel execution
bool correct_output( int *data, const int n, const int x)
{
    for(int i = 0; i < n; i++)
        if(data[i] !=x)
        {
            printf("Error! data[%d] = %d, ref = %d\n", i, data[i], x);
            return false;
        }
    return true;
}

int main(int argc, char *argv[])
{
    printf("[%s] - Starting..\n", argv[0]);
    
    // parameter setting to use persistent thread
    //int n = 30  * 20 * 1024;              // i should handle overflow prob.
    int n = 30 * 20 * 16;   
    
    int nbytes = n * sizeof(int);
    int value = 10;

    const int num_sm = 30;
    
    // allocate host memory

    int *a = 0;                             // 1. input array
    hipHostMalloc((void**)&a, nbytes);
    memset(a, 0, nbytes);

    int host_max_blocks;                    // 2. set integer variable to memcpy to 'max_blocks'
    host_max_blocks = 5;

    int totalTask;                          // "slateMax" : total number of CTA

    // allocate device memory
    int *d_a=0;                             // 1. output array
    hipMalloc((void**)&d_a, nbytes);
    hipMemset(d_a, 255, nbytes);

    int *block_index = 0;                   // 2. SM usage reporting array
    hipMalloc((void**)&block_index, sizeof(int) * (num_sm));
    hipMemset(block_index, 0, sizeof(int) * (num_sm));

    int *max_blocks = 0;                    // 3. to let device know number of Maximum blocks that SM can host
    hipMalloc((void**)&max_blocks, sizeof(int));
    hipMemset(max_blocks, 0, sizeof(int));


    // set kernel launch configuration
    //dim3 threads = dim3(32, 32 ,1);
    dim3 threads = dim3(4, 4, 1);
    dim3 blocks =  dim3(30 ,20, 1);
    
    totalTask = blocks.x * blocks.y * blocks.z;

    // create cuda event handles
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipDeviceSynchronize();
    float gpu_time = 0.0f;
    
    // original execution
    hipEventRecord(start, 0);
    hipMemcpy(d_a, a, nbytes, hipMemcpyHostToDevice);
    incremental_kernel_2d<<<blocks, threads>>>(d_a, value);
    hipMemcpy(a, d_a, nbytes, hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);

    // print the cpu and gpu times
    printf("time spent executing original kernel: %.2f\n", gpu_time);
  
    // check the output of correctness 
    bool bFinalResults = correct_output(a, n, value);
    
    ///////////////////////////////////////////////////////////////
    
    memset(a, 0, nbytes);
    hipMemset(d_a, 255, nbytes);
    hipMemcpy(d_a, a, nbytes, hipMemcpyHostToDevice);

    gpu_time = 0.0f;
    hipEventRecord(start, 0);
    hipMemcpy(d_a, a, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(max_blocks, &host_max_blocks, sizeof(int), hipMemcpyHostToDevice);

    int currentIdx = 0;
    int start_sm = 6;
    int end_sm = 15;
    // dispatch part
    do {
        TransformedKernel<<<blocks, threads>>>( start_sm, end_sm,
                                                d_a, value,
                                                totalTask,
                                                block_index, max_blocks);        
        hipMemcpyFromSymbol(&currentIdx, HIP_SYMBOL(taskIdx), sizeof(taskIdx), 0, hipMemcpyDeviceToHost);
    }while( currentIdx < totalTask);
                                           
    hipMemcpy(a, d_a, nbytes, hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);

    printf("time spent executing second kernel: %.2f\n", gpu_time);
    
    // check the output for correctness
    bool bFinalResults2 = correct_output(a, n, value);

    // release resources
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipHostFree(a);
    hipFree(d_a);
    hipFree(block_index);
    hipFree(max_blocks);

    exit(bFinalResults2 ? EXIT_SUCCESS : EXIT_FAILURE);
}
